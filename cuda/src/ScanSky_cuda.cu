#include "hip/hip_runtime.h"
/*
* Contar cuerpos celestes
*
* Asignatura Computación Paralela (Grado Ingeniería Informática)
* Código CUDA
*
* @author Ana Moretón Fernández,
* @author Arturo Gonzalez-Escribano
* @author Sergio García Prado (@garciparedes)
* @author Adrian Calvo Rojo
* @version v3.0
*
* (c) 2017, Grupo Trasgo, Universidad de Valladolid
*/

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cputils.h"


#define rowsBloqShape 8
#define columnsBloqShape 16


/*
*
* CUDA MEMCHECK
* code from: http://www.orangeowlsolutions.com/archives/613
*/
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
    }
}


__device__ __constant__ int rowsDevice;
__device__ __constant__ int columnsDevice;
__device__ __constant__ char* matrixDataPointer;
__device__ int numBlocksDevice;
__device__ char flagCambioDevice;

__global__ void kernelFillMatrixResult(int *matrixResult, int *matrixResultCopy) {

    const int ij = (blockIdx.y * blockDim.y + threadIdx.y)*columnsDevice +
                        blockIdx.x * blockDim.x + threadIdx.x;

	if(ij > -1 && ij<rowsDevice*columnsDevice){
		if(matrixDataPointer[ij] !=0){
			matrixResult[ij]=ij;
			matrixResultCopy[ij]=ij;
		} else {
			matrixResult[ij]=-1;
			matrixResultCopy[ij]=-1;
		}
	}
}

__global__ void kernelComputationLoop(int *matrixResult,int *matrixResultCopy) {

	const int i = blockIdx.y * blockDim.y + threadIdx.y;
	const int j = blockIdx.x * blockDim.x + threadIdx.x;

	/* 4.2.2 Computo y detecto si ha habido cambios */
	if(i > 0 && i<rowsDevice-1 &&
		j > 0 && j<columnsDevice-1){

		if(matrixResult[i*columnsDevice+j] != -1){

			matrixResult[i*columnsDevice+j] = matrixResultCopy[i*columnsDevice+j];
			if((matrixDataPointer[(i-1)*columnsDevice+j] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[(i-1)*columnsDevice+j]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[(i-1)*columnsDevice+j];
				flagCambioDevice = 1;
			}
			if((matrixDataPointer[(i+1)*columnsDevice+j] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[(i+1)*columnsDevice+j]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[(i+1)*columnsDevice+j];
				flagCambioDevice = 1;
			}
			if((matrixDataPointer[i*columnsDevice+j-1] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[i*columnsDevice+j-1]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[i*columnsDevice+j-1];
				flagCambioDevice = 1;
			}
			if((matrixDataPointer[i*columnsDevice+j+1] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[i*columnsDevice+j+1]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[i*columnsDevice+j+1];
				flagCambioDevice = 1;
			}
		}
	}
}

__global__ void kernelCountFigures(int *matrixResult) {

	const int i = blockIdx.y * blockDim.y + threadIdx.y;
 	const int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i > 0 && i<rowsDevice-1 &&
		j > 0 && j<columnsDevice-1 &&
			matrixResult[i*columnsDevice+j] == i*columnsDevice+j) {
				atomicAdd(&numBlocksDevice, 1);
	}
}

/**
* Funcion principal
*/
int main (int argc, char* argv[])
{

	/* 1. Leer argumento y declaraciones */
	if (argc < 2) 	{
		printf("Uso: %s <imagen_a_procesar>\n", argv[0]);
		return(EXIT_SUCCESS);
	}
	char* image_filename = argv[1];

	int rows=-1;
	int columns =-1;
	int *matrixData=NULL;
	char *matrixDataChar=NULL;
	int *matrixResult=NULL;
	int *matrixResultCopy=NULL;
	int numBlocks=-1;
	int t=-1;
	char flagCambio=-1;

	char *matrixDataDevice;
	int *matrixResultDevice;
	int *matrixResultCopyDevice;
	int *temp;

	/* 2. Leer Fichero de entrada e inicializar datos */

	/* 2.1 Abrir fichero */
	FILE *f = cp_abrir_fichero(image_filename);

	// Compruebo que no ha habido errores
	if (f==NULL)
	{
	   perror ("Error al abrir fichero.txt");
	   return -1;
	}

	/* 2.2 Leo valores del fichero */
	int i,j;
	fscanf (f, "%d\n", &rows);
	fscanf (f, "%d\n", &columns);
	// Añado dos filas y dos columnas mas para los bordes
	rows=rows+2;
	columns = columns+2;

	/* 2.3 Reservo la memoria necesaria para la matriz de datos */
	matrixData= (int *)malloc( rows*(columns) * sizeof(int) );
	if ( (matrixData == NULL)   ) {
 		perror ("Error reservando memoria");
	   	return -1;
	}

	/* 2.4 Inicializo matrices */
	for(i=0;i< rows; i++){
		for(j=0;j< columns; j++){
			matrixData[i*(columns)+j]=-1;
		}
	}
	/* 2.5 Relleno bordes de la matriz */
	for(i=1;i<rows-1;i++){
		matrixData[i*(columns)+0]=0;
		matrixData[i*(columns)+columns-1]=0;
	}
	for(i=1;i<columns-1;i++){
		matrixData[0*(columns)+i]=0;
		matrixData[(rows-1)*(columns)+i]=0;
	}
	/* 2.6 Relleno la matriz con los datos del fichero */
	for(i=1;i<rows-1;i++){
		for(j=1;j<columns-1;j++){
			fscanf (f, "%d\n", &matrixData[i*(columns)+j]);
		}
	}
	fclose(f);

	#ifdef WRITE
		printf("Inicializacion \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixData[i*(columns)+j]);
			}
			printf("\n");
		}
	#endif

	gpuErrorCheck(hipSetDevice(0));
	gpuErrorCheck(hipDeviceSynchronize());

	/* PUNTO DE INICIO MEDIDA DE TIEMPO */
	double t_ini = cp_Wtime();

//
// EL CODIGO A PARALELIZAR COMIENZA AQUI
//


	const dim3 bloqShapeGpu(columnsBloqShape,rowsBloqShape);
	const dim3 gridShapeGpu(
		ceil((float) columns / columnsBloqShape),
		ceil((float) rows / rowsBloqShape)
	);

	size_t pitch1,pitch2,pitch3;

	gpuErrorCheck(hipMallocPitch(&matrixResultDevice, &pitch1, rows*sizeof(int), columns));
	gpuErrorCheck(hipMallocPitch(&matrixResultCopyDevice, &pitch2, rows*sizeof(int), columns));
	gpuErrorCheck(hipMallocPitch(&matrixDataDevice, &pitch3, rows*sizeof(char), columns));
	//gpuErrorCheck(hipMalloc(&matrixDataDevice, sizeof(char) * rows * columns));

	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(rowsDevice),&rows, sizeof(int),0,hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(columnsDevice),&columns, sizeof(int),0,hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrixDataPointer),&matrixDataDevice, sizeof(char *)));

	//gpuErrorCheck(hipHostMalloc(&matrixDataChar, rows*(columns) * sizeof(char)));
	matrixDataChar= (char *)malloc( rows*(columns) * sizeof(char) );
	for(i = 0; i < rows * columns; i++){
		matrixDataChar[i] = matrixData[i];
	}
	/*
	gpuErrorCheck(hipMemcpy2D(
		matrixDataDevice,
		pitch3,
		matrixDataChar,
		rows*sizeof(char),
		rows*sizeof(char),
		columns,
		hipMemcpyHostToDevice
	));
	*/
	gpuErrorCheck(hipMemcpyAsync(matrixDataDevice,matrixDataChar, sizeof(char) * rows * columns,hipMemcpyHostToDevice));


	/* 3. Etiquetado inicial */


	kernelFillMatrixResult<<<gridShapeGpu, bloqShapeGpu>>>(matrixResultDevice,
		matrixResultCopyDevice);
	gpuErrorCheck(hipPeekAtLastError());

	/* 4. Computacion */
	t=0;
	/* 4.1 Flag para ver si ha habido cambios y si se continua la ejecucion */
	flagCambio=1;

	for(t=0; flagCambio != 0; t++){

		flagCambio = 0;
		gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(flagCambioDevice),&flagCambio, sizeof(char),0,hipMemcpyHostToDevice));

		temp = matrixResultDevice;
		matrixResultDevice = matrixResultCopyDevice;
		matrixResultCopyDevice = temp;

		kernelComputationLoop<<<gridShapeGpu, bloqShapeGpu>>>(matrixResultDevice,
			matrixResultCopyDevice);
		gpuErrorCheck(hipPeekAtLastError());
		gpuErrorCheck(hipMemcpyFromSymbolAsync(&flagCambio, HIP_SYMBOL(flagCambioDevice), sizeof(char), 0, hipMemcpyDeviceToHost));
	}


	/* 4.3 Inicio cuenta del numero de bloques */
	numBlocks = 0;
	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(numBlocksDevice),&numBlocks, sizeof(int),0,hipMemcpyHostToDevice));

	kernelCountFigures<<<gridShapeGpu, bloqShapeGpu>>>(matrixResultDevice);
	gpuErrorCheck(hipPeekAtLastError());

	gpuErrorCheck(hipMemcpyFromSymbolAsync(&numBlocks, HIP_SYMBOL(numBlocksDevice), sizeof(int), 0, hipMemcpyDeviceToHost));

//
// EL CODIGO A PARALELIZAR TERMINA AQUI
//

	/* PUNTO DE FINAL DE MEDIDA DE TIEMPO */
	gpuErrorCheck(hipDeviceSynchronize());
 	double t_fin = cp_Wtime();


	/* 5. Comprobación de resultados */
  	double t_total = (double)(t_fin - t_ini);

	printf("Result: %d:%d\n", numBlocks, t);
	printf("Time: %lf\n", t_total);
	#ifdef WRITE
		printf("Resultado: \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixResult[i*columns+j]);
			}
			printf("\n");
		}
	#endif

	/* 6. Liberacion de memoria */
	free(matrixData);
	free(matrixResult);
	free(matrixResultCopy);

	/*Liberamos memoria del DEVICE*/
	gpuErrorCheck(hipFree(matrixDataPointer));
	gpuErrorCheck(hipFree(matrixDataDevice));
	gpuErrorCheck(hipFree(matrixResultDevice));
	gpuErrorCheck(hipFree(matrixResultCopyDevice));

	/*Liberamos los hilos del DEVICE*/
	gpuErrorCheck(hipDeviceReset());
	return 0;
}
