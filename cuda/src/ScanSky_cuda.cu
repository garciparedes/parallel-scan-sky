#include "hip/hip_runtime.h"
/*
* Contar cuerpos celestes
*
* Asignatura Computación Paralela (Grado Ingeniería Informática)
* Código CUDA
*
* @author Ana Moretón Fernández,
* @author Arturo Gonzalez-Escribano
* @author Sergio García Prado (@garciparedes)
* @author Adrian Calvo Rojo
* @version v3.0
*
* (c) 2017, Grupo Trasgo, Universidad de Valladolid
*/

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cputils.h"


#define rowsBloqShape 8
#define columnsBloqShape 16
#define nStreams 4

/*
*
* CUDA MEMCHECK
* code from: http://www.orangeowlsolutions.com/archives/613
*/
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
    }
}


__device__ __constant__ int rowsDevice;
__device__ __constant__ int columnsDevice;
__device__ __constant__ char* matrixDataPointer;
__device__ int numBlocksDevice;

__global__ void kernelFillMatrixResult(int *matrixResult) {

    const int ij = (blockIdx.y * blockDim.y + threadIdx.y)*columnsDevice +
                        blockIdx.x * blockDim.x + threadIdx.x;

	if(ij > -1 && ij<rowsDevice*columnsDevice){
		if(matrixDataPointer[ij] !=0){
            matrixResult[ij]=ij;
		} else {
            matrixResult[ij]=-1;
		}
	}
}

__global__ void kernelComputationLoop(int *matrixResult,int *matrixResultCopy, char *flagCambioDevice) {

	const int i = blockIdx.y * blockDim.y + threadIdx.y+1;
	const int j = blockIdx.x * blockDim.x + threadIdx.x+1;

	/* 4.2.2 Computo y detecto si ha habido cambios */
	if(i > 0 && i<rowsDevice-1 &&
		j > 0 && j<columnsDevice-1){

		if(matrixResult[i*columnsDevice+j] != -1){

			matrixResult[i*columnsDevice+j] = matrixResultCopy[i*columnsDevice+j];
			if((matrixDataPointer[(i-1)*columnsDevice+j] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[(i-1)*columnsDevice+j]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[(i-1)*columnsDevice+j];
				flagCambioDevice[0] = 1;
			}
			if((matrixDataPointer[(i+1)*columnsDevice+j] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[(i+1)*columnsDevice+j]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[(i+1)*columnsDevice+j];
				flagCambioDevice[0] = 1;
			}
			if((matrixDataPointer[i*columnsDevice+j-1] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[i*columnsDevice+j-1]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[i*columnsDevice+j-1];
				flagCambioDevice[0] = 1;
			}
			if((matrixDataPointer[i*columnsDevice+j+1] == matrixDataPointer[i*columnsDevice+j]) &&
				(matrixResult[i*columnsDevice+j] > matrixResultCopy[i*columnsDevice+j+1]))
			{
				matrixResult[i*columnsDevice+j] = matrixResultCopy[i*columnsDevice+j+1];
				flagCambioDevice[0] = 1;
			}
		}
	}
}

__global__ void kernelCountFigures(int *matrixResult) {

	const int i = blockIdx.y * blockDim.y + threadIdx.y+1;
 	const int j = blockIdx.x * blockDim.x + threadIdx.x+1;

	if(i > 0 && i<rowsDevice-1 &&
		j > 0 && j<columnsDevice-1 &&
			matrixResult[i*columnsDevice+j] == i*columnsDevice+j) {
				atomicAdd(&numBlocksDevice, 1);
	}
}

/**
* Funcion principal
*/
int main (int argc, char* argv[])
{
    const char zero = 0;

	/* 1. Leer argumento y declaraciones */
	if (argc < 2) 	{
		printf("Uso: %s <imagen_a_procesar>\n", argv[0]);
		return(EXIT_SUCCESS);
	}
	char* image_filename = argv[1];

	int rows=-1;
	int columns =-1;
	int *matrixData=NULL;
	char *matrixDataChar=NULL;
	int *matrixResult=NULL;
	int *matrixResultCopy=NULL;
	int numBlocks=-1;
	int t=-1;
	char flagCambio=-1;

	char *matrixDataDevice;
	int *matrixResult1Device;
    int *matrixResult2Device;
    int *matrixResult3Device;
    int *matrixResult4Device;
    char *flagCambioDevice1;
    char *flagCambioDevice2;
    char *flagCambioDevice3;
    char *flagCambioDevice4;

	/* 2. Leer Fichero de entrada e inicializar datos */

	/* 2.1 Abrir fichero */
	FILE *f = cp_abrir_fichero(image_filename);

	// Compruebo que no ha habido errores
	if (f==NULL)
	{
	   perror ("Error al abrir fichero.txt");
	   return -1;
	}

	/* 2.2 Leo valores del fichero */
	int i,j;
	fscanf (f, "%d\n", &rows);
	fscanf (f, "%d\n", &columns);
	// Añado dos filas y dos columnas mas para los bordes
	rows=rows+2;
	columns = columns+2;

	/* 2.3 Reservo la memoria necesaria para la matriz de datos */
	matrixData= (int *)malloc( rows*(columns) * sizeof(int) );
	if ( (matrixData == NULL)   ) {
 		perror ("Error reservando memoria");
	   	return -1;
	}

	/* 2.4 Inicializo matrices */
	for(i=0;i< rows; i++){
		for(j=0;j< columns; j++){
			matrixData[i*(columns)+j]=-1;
		}
	}
	/* 2.5 Relleno bordes de la matriz */
	for(i=1;i<rows-1;i++){
		matrixData[i*(columns)+0]=0;
		matrixData[i*(columns)+columns-1]=0;
	}
	for(i=1;i<columns-1;i++){
		matrixData[0*(columns)+i]=0;
		matrixData[(rows-1)*(columns)+i]=0;
	}
	/* 2.6 Relleno la matriz con los datos del fichero */
	for(i=1;i<rows-1;i++){
		for(j=1;j<columns-1;j++){
			fscanf (f, "%d\n", &matrixData[i*(columns)+j]);
		}
	}
	fclose(f);

	#ifdef WRITE
		printf("Inicializacion \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixData[i*(columns)+j]);
			}
			printf("\n");
		}
	#endif

	gpuErrorCheck(hipSetDevice(0));
	gpuErrorCheck(hipDeviceSynchronize());

	/* PUNTO DE INICIO MEDIDA DE TIEMPO */
	double t_ini = cp_Wtime();

//
// EL CODIGO A PARALELIZAR COMIENZA AQUI
//
  hipStream_t stream[4];
  gpuErrorCheck( hipStreamCreate(&stream[0]) );
  gpuErrorCheck( hipStreamCreate(&stream[1]) );
  gpuErrorCheck( hipStreamCreate(&stream[2]) );
  gpuErrorCheck( hipStreamCreate(&stream[3]) );

	const dim3 bloqShapeGpu(columnsBloqShape,rowsBloqShape);
	const dim3 gridShapeGpu(
		ceil((float) columns / columnsBloqShape),
		ceil((float) rows / rowsBloqShape)
	);

	size_t pitch,pitch3;

  const dim3 gridShapeGpuMin(
      ceil((float) (columns-1) / columnsBloqShape),
      ceil((float) (rows-1) / rowsBloqShape)
  );

  gpuErrorCheck(hipMalloc(&flagCambioDevice1, sizeof(char)));
  gpuErrorCheck(hipMalloc(&flagCambioDevice2, sizeof(char)));
  gpuErrorCheck(hipMalloc(&flagCambioDevice3, sizeof(char)));
  gpuErrorCheck(hipMalloc(&flagCambioDevice4, sizeof(char)));
  gpuErrorCheck(hipMallocPitch(&matrixResult1Device, &pitch, rows*sizeof(int), columns));
  gpuErrorCheck(hipMallocPitch(&matrixResult2Device, &pitch, rows*sizeof(int), columns));
  gpuErrorCheck(hipMallocPitch(&matrixResult3Device, &pitch, rows*sizeof(int), columns));
  gpuErrorCheck(hipMallocPitch(&matrixResult4Device, &pitch, rows*sizeof(int), columns));
	gpuErrorCheck(hipMallocPitch(&matrixDataDevice, &pitch3, rows*sizeof(char), columns));
	//gpuErrorCheck(hipMalloc(&matrixDataDevice, sizeof(char) * rows * columns));

	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(rowsDevice),&rows, sizeof(int),0,hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(columnsDevice),&columns, sizeof(int),0,hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrixDataPointer),&matrixDataDevice, sizeof(char *)));

	//gpuErrorCheck(hipHostMalloc(&matrixDataChar, rows*(columns) * sizeof(char)));
	matrixDataChar= (char *)malloc( rows*(columns) * sizeof(char) );
	for(i = 0; i < rows * columns; i++){
		matrixDataChar[i] = matrixData[i];
	}
	/*
	gpuErrorCheck(hipMemcpy2D(
		matrixDataDevice,
		pitch3,
		matrixDataChar,
		rows*sizeof(char),
		rows*sizeof(char),
		columns,
		hipMemcpyHostToDevice
	));
	*/
	gpuErrorCheck(hipMemcpyAsync(matrixDataDevice,matrixDataChar,
        sizeof(char) * rows * columns,hipMemcpyHostToDevice));


	/* 3. Etiquetado inicial */


	kernelFillMatrixResult<<<gridShapeGpu, bloqShapeGpu>>>(matrixResult4Device);
	gpuErrorCheck(hipPeekAtLastError());

	/* 4. Computacion */
	t=0;
	/* 4.1 Flag para ver si ha habido cambios y si se continua la ejecucion */
	flagCambio=1;

  gpuErrorCheck(hipMemcpyAsync(flagCambioDevice1,&zero, sizeof(char),
      hipMemcpyHostToDevice,stream[0]));
  kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[0]>>>(
      matrixResult1Device,matrixResult4Device,flagCambioDevice1);

  gpuErrorCheck(hipMemcpyAsync(flagCambioDevice2,&zero, sizeof(char),
      hipMemcpyHostToDevice,stream[0]));
  kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[0]>>>(
      matrixResult2Device,matrixResult1Device,flagCambioDevice2);

  gpuErrorCheck(hipMemcpyAsync(flagCambioDevice3,&zero, sizeof(char),
      hipMemcpyHostToDevice,stream[0]));
  kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[0]>>>(
      matrixResult3Device,matrixResult2Device,flagCambioDevice3);

  gpuErrorCheck(hipMemcpyAsync(flagCambioDevice4,&zero, sizeof(char),
      hipMemcpyHostToDevice,stream[0]));
  kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[0]>>>(
      matrixResult4Device,matrixResult3Device,flagCambioDevice4);

  for(t=0; flagCambio != 0; t++){
  flagCambio = 0;
      if(t % nStreams == 0){
          gpuErrorCheck(hipMemcpyAsync(&flagCambio,flagCambioDevice1, sizeof(char),
              hipMemcpyDeviceToHost,stream[0]));
          gpuErrorCheck(hipMemcpyAsync(flagCambioDevice1,&zero, sizeof(char),
              hipMemcpyHostToDevice,stream[0]));
          kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[0]>>>(
              matrixResult1Device,matrixResult4Device,flagCambioDevice1);
      } else if(t % nStreams == 1){
          gpuErrorCheck(hipMemcpyAsync(&flagCambio,flagCambioDevice2, sizeof(char),
              hipMemcpyDeviceToHost,stream[1]));
          gpuErrorCheck(hipMemcpyAsync(flagCambioDevice2,&zero, sizeof(char),
              hipMemcpyHostToDevice,stream[1]));
          kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[1]>>>(
              matrixResult2Device,matrixResult1Device,flagCambioDevice2);
      } else if(t % nStreams == 2){
          gpuErrorCheck(hipMemcpyAsync(&flagCambio,flagCambioDevice3, sizeof(char),
              hipMemcpyDeviceToHost,stream[2]));
          gpuErrorCheck(hipMemcpyAsync(flagCambioDevice3,&zero, sizeof(char),
              hipMemcpyHostToDevice,stream[2]));
          kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[2]>>>(
              matrixResult3Device,matrixResult2Device,flagCambioDevice3);
      } else {
          gpuErrorCheck(hipMemcpyAsync(&flagCambio,flagCambioDevice4, sizeof(char),
              hipMemcpyDeviceToHost,stream[3]));
          gpuErrorCheck(hipMemcpyAsync(flagCambioDevice4,&zero, sizeof(char),
              hipMemcpyHostToDevice,stream[3]));
          kernelComputationLoop<<<gridShapeGpuMin, bloqShapeGpu,0,stream[3]>>>(
              matrixResult4Device,matrixResult3Device,flagCambioDevice4);
      }
	}


	/* 4.3 Inicio cuenta del numero de bloques */
	numBlocks = 0;
	gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(numBlocksDevice),&numBlocks,
        sizeof(int),0,hipMemcpyHostToDevice));
    if(t % nStreams == 0){
        kernelCountFigures<<<gridShapeGpuMin, bloqShapeGpu>>>(matrixResult1Device);
    } else if(t % nStreams == 1){
        kernelCountFigures<<<gridShapeGpuMin, bloqShapeGpu>>>(matrixResult2Device);
    } else if(t % nStreams == 2){
        kernelCountFigures<<<gridShapeGpuMin, bloqShapeGpu>>>(matrixResult3Device);
    } else {
        kernelCountFigures<<<gridShapeGpuMin, bloqShapeGpu>>>(matrixResult4Device);
    }

	gpuErrorCheck(hipPeekAtLastError());

	gpuErrorCheck(hipMemcpyFromSymbolAsync(&numBlocks, HIP_SYMBOL(numBlocksDevice),
        sizeof(int), 0, hipMemcpyDeviceToHost));

//
// EL CODIGO A PARALELIZAR TERMINA AQUI
//

	/* PUNTO DE FINAL DE MEDIDA DE TIEMPO */
	gpuErrorCheck(hipDeviceSynchronize());
 	double t_fin = cp_Wtime();


	/* 5. Comprobación de resultados */
  	double t_total = (double)(t_fin - t_ini);

	printf("Result: %d:%d\n", numBlocks, t);
	printf("Time: %lf\n", t_total);
	#ifdef WRITE
		printf("Resultado: \n");
		for(i=0;i<rows;i++){
			for(j=0;j<columns;j++){
				printf ("%d\t", matrixResult[i*columns+j]);
			}
			printf("\n");
		}
	#endif

	/* 6. Liberacion de memoria */
	free(matrixData);
	free(matrixResult);
	free(matrixResultCopy);

	/*Liberamos memoria del DEVICE*/
	gpuErrorCheck(hipFree(matrixDataPointer));
	gpuErrorCheck(hipFree(matrixDataDevice));
	gpuErrorCheck(hipFree(matrixResult1Device));
    gpuErrorCheck(hipFree(matrixResult2Device));
    gpuErrorCheck(hipFree(matrixResult3Device));
    gpuErrorCheck(hipFree(matrixResult4Device));
    gpuErrorCheck( hipStreamDestroy(stream[0]) );
    gpuErrorCheck( hipStreamDestroy(stream[1]) );
    gpuErrorCheck( hipStreamDestroy(stream[2]) );
    gpuErrorCheck( hipStreamDestroy(stream[3]) );

	/*Liberamos los hilos del DEVICE*/
	gpuErrorCheck(hipDeviceReset());
	return 0;
}
