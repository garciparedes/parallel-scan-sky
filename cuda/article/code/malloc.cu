gpuErrorCheck(hipMalloc(&flagCambio_d, sizeof(char) * nStreams));
gpuErrorCheck(hipMallocPitch(&matrixResult_d, &pitch, rows*sizeof(int), columns * nStreams));
gpuErrorCheck(hipMallocPitch(&matrixDataChar_d, &pitch3, rows*sizeof(char), columns));
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(rows_d),&rows, sizeof(int),0,hipMemcpyHostToDevice));
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(columns_d),&columns, sizeof(int),0,hipMemcpyHostToDevice));
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrixData_d),&matrixDataChar_d, sizeof(char *)));
matrixDataChar= (char *)malloc( rows*(columns) * sizeof(char) );
for(i = 0; i < rows * columns; i++){
    matrixDataChar[i] = matrixData[i];
}
gpuErrorCheck(hipMemcpyAsync(matrixDataChar_d,matrixDataChar,
    sizeof(char) * rows * columns,hipMemcpyHostToDevice));
