gpuErrorCheck(hipMalloc(&flagCambio_d_0, sizeof(char)));
// ...
gpuErrorCheck(hipMallocPitch(&matrixResult_d_0, &pitch, rows*sizeof(int), columns));
// ...
gpuErrorCheck(hipMallocPitch(&matrixData_d, &pitch3, rows*sizeof(char), columns));
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(rows_d),&rows, sizeof(int),0,hipMemcpyHostToDevice));
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(columns_d),&columns, sizeof(int),0,hipMemcpyHostToDevice));
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrixData_d),&matrixDataChar_d, sizeof(char *)));
matrixDataChar = (char *)malloc( rows * columns * sizeof(char) );
for(i = 0; i < rows * columns; i++){
    matrixDataChar[i] = matrixData[i];
}
gpuErrorCheck(hipMemcpyAsync(matrixDataChar_d,matrixDataChar,
    sizeof(char) * rows * columns, hipMemcpyHostToDevice));
