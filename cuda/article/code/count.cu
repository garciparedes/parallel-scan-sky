numBlocks = 0;
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(numBlocks_d),&numBlocks,
    sizeof(int),0,hipMemcpyHostToDevice));
if(t % nStreams == 0){
    kernelCountFigures<<<gridShapeGpu, bloqShapeGpu>>>(matrixResult_d_0);
} else if(t % nStreams == 1){
    // ...
} // ...
gpuErrorCheck(hipPeekAtLastError());
gpuErrorCheck(hipMemcpyFromSymbolAsync(&numBlocks, HIP_SYMBOL(numBlocks_d),
    sizeof(int), 0, hipMemcpyDeviceToHost));
