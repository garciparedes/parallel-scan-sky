numBlocks = 0;
gpuErrorCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(numBlocks_d),&numBlocks,
    sizeof(int),0,hipMemcpyHostToDevice));
kernelCountFigures<<<gridShapeGpu, bloqShapeGpu>>>(
    &matrixResult_d[rows * columns * (t % nStreams)]
);
gpuErrorCheck(hipPeekAtLastError());
gpuErrorCheck(hipMemcpyFromSymbolAsync(&numBlocks, HIP_SYMBOL(numBlocks_d),
    sizeof(int), 0, hipMemcpyDeviceToHost));
